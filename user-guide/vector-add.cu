/*
 *   Vector addition example using CUDA.
 *   This is a non-optimised example that is likely to benefit
 *   from
 *   - adaptig the main kernel launch configuration, so that
 *     it creates a grid containing a number of blocks that is
 *     a multiple of the number of SMs on the device.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Host function to initialize input vector elements. This
 * function simply initializes each element in the vector to
 * a constant number 'num'.
 */

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of
 * corresponding elements in input vectors `a` and
 * `b`. Note that the function assumes `a` and `b`
 * are of the same size N.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("All values were calculated correctly. Well done.\n");
}

int main()
{
  const int N = 50000000;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(7, b, N);
  initWith(0, c, N);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 32;
  numberOfBlocks = 32;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(10, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
